
#include <hip/hip_runtime.h>
extern "C" __global__ void
velocity_one(float2* psi1, float2* psi2, int resy, int resz, int num, float hbar, float pi, float* vx, float* vy, float* vz)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    float2 c1 = make_float2(psi1[i].x, -psi1[i].y);
    float2 c2 = make_float2(psi2[i].x, -psi2[i].y);
    
    float2 mul1 = psi1[(i / resz) * resz + (i + 1) % resz];
    float2 mul2 = psi2[(i / resz) * resz + (i + 1) % resz];
    float2 summ = make_float2((c1.x * mul1.x - c1.y * mul1.y + c2.x * mul2.x - c2.y * mul2.y), 
                             (c1.x * mul1.y + c1.y * mul1.x + c2.x * mul2.y + c2.y * mul2.x));
    float result = (float)atan2(summ.y, summ.x);
    if (abs(summ.y) < 0.00001) {
        result *= -1;
    }
    vz[i] = (float)result * hbar;
                            
    mul1 = psi1[i - ((i / resz) % resy) * resz + (((i + resz) / resz) % resy) * resz];
    mul2 = psi2[i - ((i / resz) % resy) * resz + (((i + resz) / resz) % resy) * resz];
    summ = make_float2((c1.x * mul1.x - c1.y * mul1.y + c2.x * mul2.x - c2.y * mul2.y), 
                      (c1.x * mul1.y + c1.y * mul1.x + c2.x * mul2.y + c2.y * mul2.x));
    result = (float)atan2(summ.y, summ.x);
    if (abs(summ.y) < 0.00001) {
            result *= -1;
    }
    vy[i] = (float)result * hbar;
    
    mul1 = psi1[(i + resz * resy) % num];
    mul2 = psi2[(i + resz * resy) % num];
    summ = make_float2((c1.x * mul1.x - c1.y * mul1.y + c2.x * mul2.x - c2.y * mul2.y), 
                      (c1.x * mul1.y + c1.y * mul1.x + c2.x * mul2.y + c2.y * mul2.x));
    result = (float)atan2(summ.y, summ.x);
    if (abs(summ.y) < 0.00001) {
            result *= -1;
    }
    vx[i] = result * hbar;
}